#include <hip/hip_runtime.h>


__global__ void copy_kernel(float *src, float *dst, int width, int height, int stride_x_src, int stride_y_src, int stride_x_dst, int stride_y_dst) {
    const int block_size = blockDim.x;
    const int tid = threadIdx.x;
    const int element_per_thread = width * height / block_size;
    for (int i = 0; i < element_per_thread; i++) {
        const int src_x = (i * block_size + tid) % width;
        const int src_y = (i * block_size + tid) / width;
        const int dst_x = (i * block_size + tid) % width;
        const int dst_y = (i * block_size + tid) / width;
        dst[dst_y * stride_y_dst + dst_x * stride_x_dst] = src[src_y * stride_y_src + src_x * stride_x_src];
    }
}

int main() {
    const int width = 1024;
    const int height = 1024;
    const int stride_x_src = 1;
    const int stride_y_src = 1024;
    const int stride_x_dst = 1;
    const int stride_y_dst = 1024;
    float *src, *dst;
    hipMallocManaged(&src, width * height * sizeof(float));
    hipMallocManaged(&dst, width * height * sizeof(float));
    for (int i = 0; i < width * height; i++) {
        src[i] = static_cast<float>(i);
    }
    copy_kernel<<<1, 1024>>>(src, dst, width, height, stride_x_src, stride_y_src, stride_x_dst, stride_y_dst);
    if (hipGetLastError() != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(hipGetLastError()));
        return 1;
    }
    hipDeviceSynchronize();
    for (int i = 0; i < width * height; i++) {
        if (dst[i] != src[i]) {
            printf("Error: dst[%d] = %f, src[%d] = %f\n", i, dst[i], i, src[i]);
            return 1;
        }
    }
    printf("Test passed\n");
    hipFree(src);
    hipFree(dst);
    return 0;
}
