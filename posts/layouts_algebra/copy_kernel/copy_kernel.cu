#include "hip/hip_runtime.h"
// nvcc -std=c++20 -gencode arch=compute_80,code=sm_80 copy_kernel.cu -o copy_kernel -I cutlass/include && ./copy_kernel
#include <functional>
#include <iostream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>
#include <cute/tensor.hpp>

using namespace cute;

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
void check(hipError_t err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(const char* const file, const int line)
{
    hipError_t const err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

__global__ void copy_kernel_v0(float *src, float *dst, int width, int height, int stride_x_src, int stride_y_src, int stride_x_dst, int stride_y_dst) {
    const int block_size = blockDim.x;
    const int tid = threadIdx.x;
    const int element_per_thread = width * height / block_size;
    for (int i = 0; i < element_per_thread; i++) {
        const int x = (tid * element_per_thread + i) % width;
        const int y = (tid * element_per_thread + i) / width;
        dst[y * stride_y_dst + x * stride_x_dst] = src[y * stride_y_src + x * stride_x_src];
    }
}

__global__ void copy_kernel_v0_strided(float *src, float *dst, int width, int height, int stride_x_src, int stride_y_src, int stride_x_dst, int stride_y_dst) {
    const int block_size = blockDim.x;
    const int tid = threadIdx.x;
    const int element_per_thread = width * height / block_size;
    for (int i = 0; i < element_per_thread; i++) {
        const int x = (i * block_size + tid) % width;
        const int y = (i * block_size + tid) / width;
        dst[y * stride_y_dst + x * stride_x_dst] = src[y * stride_y_src + x * stride_x_src];
    }
}

template <typename SrcLayout, typename DstLayout>
__global__ void copy_kernel_v1(float *src, float *dst) {
    constexpr auto l1 = SrcLayout{};
    constexpr auto l2 = DstLayout{};
    constexpr int width = get<0>(shape(l1));
    constexpr int height = get<1>(shape(l1));
    const int block_size = blockDim.x;
    const int tid = threadIdx.x;
    const int element_per_thread = width * height / block_size;
    #pragma unroll
    for (int i = 0; i < element_per_thread; i++) {
        const int x = (tid * element_per_thread + i) % width;
        const int y = (tid * element_per_thread + i) / width;
        dst[l2(x, y)] = src[l1(x, y)];
    }
}

template <typename SrcLayout, typename DstLayout, typename FrgThr>
__global__ void copy_kernel_v2(float *src, float *dst) {
    constexpr auto l1 = SrcLayout{};
    constexpr auto l2 = DstLayout{};
    constexpr auto frgthr = FrgThr{};
    const int tid = threadIdx.x;
    #pragma unroll
    for (int i = 0; i < size<0>(frgthr); i++) {
        dst[l2(frgthr(i, tid))] = src[l1(frgthr(i, tid))];
    }
}

enum struct Flavor {
    TILE,
    STRIDE,
    LAYOUT_TILE,
    FRGTHR,
    FRGTHR_STRIDE
};

#define FLAVOR_SWITCH(flavor, CONST_NAME, ...)                     \
[&] {                                                           \
    if (flavor == Flavor::TILE)                               \
    {                                                           \
        constexpr static Flavor CONST_NAME = Flavor::TILE;    \
        return __VA_ARGS__();                                   \
    }                                                           \
    else if (flavor == Flavor::LAYOUT_TILE)                          \
    {                                                           \
        constexpr static Flavor CONST_NAME = Flavor::LAYOUT_TILE;    \
        return __VA_ARGS__();                                   \
    }                                                           \
    else if (flavor == Flavor::FRGTHR)                          \
    {                                                           \
        constexpr static Flavor CONST_NAME = Flavor::FRGTHR;    \
        return __VA_ARGS__();                                   \
    }                                                           \
    else if (flavor == Flavor::STRIDE)                          \
    {                                                           \
        constexpr static Flavor CONST_NAME = Flavor::STRIDE;    \
        return __VA_ARGS__();                                   \
    }                                                           \
    else if (flavor == Flavor::FRGTHR_STRIDE)                   \
    {                                                           \
        constexpr static Flavor CONST_NAME = Flavor::FRGTHR_STRIDE; \
        return __VA_ARGS__();                                   \
    }                                                           \
    else                                                        \
    {                                                           \
        std::cerr << "Unsupported flavor"         << std::endl; \
        std::exit(EXIT_FAILURE);                                \
    }                                                           \
}()

#define SIZE_SWITCH(size, CONST_NAME, ...)                      \
[&] {                                                           \
    if (size == 8)                                             \
    {                                                           \
        constexpr static int CONST_NAME = 8;                   \
        return __VA_ARGS__();                                   \
    }                                                           \
    else if (size == 16)                                        \
    {                                                           \
        constexpr static int CONST_NAME = 16;                  \
        return __VA_ARGS__();                                   \
    }                                                           \
    else if (size == 32)                                        \
    {                                                           \
        constexpr static int CONST_NAME = 32;                  \
        return __VA_ARGS__();                                   \
    }                                                           \
    else if (size == 64)                                        \
    {                                                           \
        constexpr static int CONST_NAME = 64;                  \
        return __VA_ARGS__();                                   \
    }                                                           \
    else if (size == 128)                                       \
    {                                                           \
        constexpr static int CONST_NAME = 128;                 \
        return __VA_ARGS__();                                   \
    }                                                           \
    else if (size == 256)                                       \
    {                                                           \
        constexpr static int CONST_NAME = 256;                 \
        return __VA_ARGS__();                                   \
    }                                                           \
    else if (size == 512)                                       \
    {                                                           \
        constexpr static int CONST_NAME = 512;                 \
        return __VA_ARGS__();                                   \
    }                                                           \
    else if (size == 1024)                                      \
    {                                                           \
        constexpr static int CONST_NAME = 1024;                \
        return __VA_ARGS__();                                   \
    }                                                           \
    else                                                        \
    {                                                           \
        std::cerr << "Unsupported size: " << size << std::endl; \
        std::exit(EXIT_FAILURE);                                \
    }                                                           \
}()

template <int width, int height, int thread_num, typename T>
void prepare(T **src, T **dst) {
    hipMallocManaged(src, width * height * sizeof(T));
    hipMallocManaged(dst, width * height * sizeof(T));
    for (int i = 0; i < width * height; i++) {
        (*src)[i] = static_cast<T>(i);
    }
    hipMemset(*dst, static_cast<T>(0), width * height * sizeof(T));
}

template <int width, int height, int thread_num, typename T>
int check_correctness(T *src, T *dst) {
    constexpr int stride_x_src = 1;
    constexpr int stride_y_src = width;
    constexpr int stride_x_dst = height;
    constexpr int stride_y_dst = 1;
    for (int x = 0; x < width; x++) {
        for (int y = 0; y < height; y++) {
            const int src_idx = y * stride_y_src + x * stride_x_src;
            const int dst_idx = y * stride_y_dst + x * stride_x_dst;
            if (dst[dst_idx] != src[src_idx]) {
                printf("Error: dst[%d, %d] = %f, src[%d, %d] = %f\n", x, y, dst[dst_idx], x, y, src[src_idx]);
                return 1;
            }
        }
    }
    return 0;
}

template <class T>
float measure_performance(std::function<void(hipStream_t)> bound_function,
                          hipStream_t stream, unsigned int num_repeats = 100,
                          unsigned int num_warmups = 100)
{
    hipEvent_t start, stop;
    float time;

    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    for (unsigned int i{0}; i < num_warmups; ++i)
    {
        bound_function(stream);
    }

    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

    CHECK_CUDA_ERROR(hipEventRecord(start, stream));
    for (unsigned int i{0}; i < num_repeats; ++i)
    {
        bound_function(stream);
    }
    CHECK_CUDA_ERROR(hipEventRecord(stop, stream));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_LAST_CUDA_ERROR();
    CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));

    float const latency{time / num_repeats};

    return latency;
}

template <int width, int height, int thread_num, Flavor flavor>
void run(float *src, float *dst, hipStream_t stream = 0) {
    constexpr int stride_x_src = 1;
    constexpr int stride_y_src = width;
    constexpr int stride_x_dst = height;
    constexpr int stride_y_dst = 1;
    if constexpr (flavor == Flavor::TILE) {
        copy_kernel_v0<<<1, thread_num, 0, stream>>>(src, dst, width, height, stride_x_src, stride_y_src, stride_x_dst, stride_y_dst);
    } else if constexpr (flavor == Flavor::LAYOUT_TILE) {
        using src_layout = decltype(make_layout(Shape<Int<width>, Int<height>>{}, Stride<Int<stride_x_src>, Int<stride_y_src>>{}));
        using dst_layout = decltype(make_layout(Shape<Int<width>, Int<height>>{}, Stride<Int<stride_x_dst>, Int<stride_y_dst>>{}));
        copy_kernel_v1<src_layout, dst_layout><<<1, thread_num, 0, stream>>>(src, dst);
    } else if constexpr (flavor == Flavor::FRGTHR) {
        using src_layout = decltype(make_layout(Shape<Int<width>, Int<height>>{}, Stride<Int<stride_x_src>, Int<stride_y_src>>{}));
        using dst_layout = decltype(make_layout(Shape<Int<width>, Int<height>>{}, Stride<Int<stride_x_dst>, Int<stride_y_dst>>{}));
        using frgthr = decltype(make_layout(Shape<Int<width * height / thread_num>, Int<thread_num>>{}));
        copy_kernel_v2<src_layout, dst_layout, frgthr><<<1, thread_num, 0, stream>>>(src, dst);
    } else if constexpr (flavor == Flavor::STRIDE) {
        copy_kernel_v0_strided<<<1, thread_num, 0, stream>>>(src, dst, width, height, stride_x_src, stride_y_src, stride_x_dst, stride_y_dst);
    } else if constexpr (flavor == Flavor::FRGTHR_STRIDE) {
        using src_layout = decltype(make_layout(Shape<Int<width>, Int<height>>{}, Stride<Int<stride_x_src>, Int<stride_y_src>>{}));
        using dst_layout = decltype(make_layout(Shape<Int<width>, Int<height>>{}, Stride<Int<stride_x_dst>, Int<stride_y_dst>>{}));
        using frgthr = decltype(make_layout(Shape<Int<width * height / thread_num>, Int<thread_num>>{}, LayoutRight{}));
        copy_kernel_v2<src_layout, dst_layout, frgthr><<<1, thread_num, 0, stream>>>(src, dst);
    }

    CHECK_LAST_CUDA_ERROR();
}

template <int width, int height, int thread_num>
void run_all_flavors(float *src, float *dst, hipStream_t stream) {
    const std::vector<std::pair<Flavor, std::string>> flavors = {
        {Flavor::TILE, "TILE"},
        {Flavor::STRIDE, "STRIDE"}, 
        {Flavor::LAYOUT_TILE, "LAYOUT_TILE"},
        {Flavor::FRGTHR, "FRGTHR"},
        {Flavor::FRGTHR_STRIDE, "FRGTHR_STRIDE"}
    };
    
    printf("Running all flavors for size %dx%d:\n", width, height);
    printf("=====================================\n");
    
    for (const auto& [flavor, name] : flavors) {
        printf("Testing %s flavor:\n", name.c_str());
        
        // Reset destination buffer
        hipMemset(dst, 0, width * height * sizeof(float));
        
        // Create bound function for performance measurement
        auto run_bound = [&](hipStream_t s) {
            FLAVOR_SWITCH(flavor, flavor_static, [&] {
                run<width, height, thread_num, flavor_static>(src, dst, s);
            });
        };
        
        // Measure performance
        float latency = measure_performance<void>(run_bound, stream);
        printf("  Average latency: %.3f ms\n", latency);
        
        // Synchronization and correctness check
        CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
        if (check_correctness<width, height, thread_num>(src, dst)) {
            printf("  Test FAILED\n");
        } else {
            printf("  Test PASSED\n");
        }
        printf("\n");
    }
}


void parse_args(int argc, char *argv[], int &size) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <size>" << std::endl;
        std::cerr << "Valid sizes: 8, 16, 32, 64, 128, 256, 512, 1024" << std::endl;
        std::exit(EXIT_FAILURE);
    }
    
    try {
        size = std::stoi(argv[1]);
    } catch (const std::exception& e) {
        std::cerr << "Invalid size: " << argv[1] << std::endl;
        std::cerr << "Valid sizes: 8, 16, 32, 64, 128, 256, 512, 1024" << std::endl;
        std::exit(EXIT_FAILURE);
    }
    
    // Validate size is one of the supported sizes
    if (size != 8 && size != 16 && size != 32 && size != 64 && 
        size != 128 && size != 256 && size != 512 && size != 1024) {
        std::cerr << "Unsupported size: " << size << std::endl;
        std::cerr << "Valid sizes: 8, 16, 32, 64, 128, 256, 512, 1024" << std::endl;
        std::exit(EXIT_FAILURE);
    }
}


int main(int argc, char *argv[]) {
    int size;
    parse_args(argc, argv, size);
    constexpr int thread_num = 32;
    float *src = nullptr, *dst = nullptr;
    hipStream_t stream;
    CHECK_CUDA_ERROR(hipStreamCreate(&stream));
    
    return SIZE_SWITCH(size, size_static, [&] {
        constexpr int width = size_static;
        constexpr int height = size_static;
        prepare<width, height, thread_num>(&src, &dst);
        
        // Run all flavors for the given size
        run_all_flavors<width, height, thread_num>(src, dst, stream);
        
        printf("All tests completed for size %dx%d\n", width, height);
        CHECK_CUDA_ERROR(hipStreamDestroy(stream));
        return 0;
    });
}